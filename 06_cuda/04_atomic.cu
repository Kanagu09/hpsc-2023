
#include <hip/hip_runtime.h>
#include <cstdio>

__device__ __managed__ int sum;

__global__ void reduction(int &sum) {
  //sum += 1;
  atomicAdd(&sum, 1);
}

int main(void) {
  const int N = 128;
  const int M = 64;
  sum = 0;
  reduction<<<N/M,M>>>(sum);
  hipDeviceSynchronize();
  printf("%d\n",sum);
}

// atomicAdd は，大量の thread から同時に global memory にアクセスが飛ぶので，thread 数によってはかなり処理が遅くなる

